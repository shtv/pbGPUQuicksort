/*
 * Copyright 2010 Pawel Baran.
 * 
 * shatov33@gmail.com
 *
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#define MAX_NUM_OF_THREADS_PER_BLOCK 1
#define MAX_NUM_OF_BLOCKS 65536

#define NUM_OF_ELEMENTS 4  // k, where k = 1, 2, ...
#define NUM_OF_ARRAYS_PER_BLOCK 6
#define MAX_SHARED_MEMORY_SIZE 

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <limits.h>

#include <time.h>

#include <cutil_inline.h>

#include <quicksort_kernel.cu>

#include "elem.h"
#include "elem.cu"

void runTest( int argc, char** argv);

/*
extern "C" 
unsigned int compare( const int* reference, const float* data, 
					  const unsigned int len);
extern "C" 
void computeGold( int* reference, int* idata, const unsigned int len);
*/

int main( int argc, char** argv){
	runTest( argc, argv);
	cutilExit(argc, argv);
}

void down_sweep_for_sum(sum* d_sums,int num_sums,int n){
	int blocks_num=num_sums/(MAX_NUM_OF_THREADS_PER_BLOCK*2);

	if(MAX_NUM_OF_THREADS_PER_BLOCK*2*blocks_num!=num_sums)
		blocks_num+=1;

	dim3 grid(blocks_num,1,1);
	int threads_num;

	if(blocks_num==1)
		threads_num=num_sums/2;
	else
		threads_num=MAX_NUM_OF_THREADS_PER_BLOCK;

	if(!threads_num) return;

	if(blocks_num>1){
		dim3 grid2(1,1,1);
		dim3 threads2(blocks_num/2,1,1);
		int offset=MAX_NUM_OF_THREADS_PER_BLOCK*2;
		printf("2:offset=%d threads to sum of sums = %d \n",offset,threads2.x);
		accumulate_sum_of_sums2<<<grid2,threads2,6*sizeof(int)*threads2.x>>> (d_sums,2,offset);
		cutilCheckMsg("accumulate_sum_of_sums2");
	}

	dim3 threads(threads_num,1,1);
	printf("second of the accumulating functions: blocks=%d threads in each one=%d\n",blocks_num,threads_num);
	accumulate_sums2<<<grid,threads,6*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK>>> (d_sums,2,blocks_num==1);
	cutilCheckMsg("accumulate_sums2");
}

void up_sweep_for_sum(sum* d_sums,int num_sums,int n){
	int blocks_num=num_sums/(MAX_NUM_OF_THREADS_PER_BLOCK*2);

	if(MAX_NUM_OF_THREADS_PER_BLOCK*2*blocks_num!=num_sums)
		blocks_num+=1;

	dim3 grid(blocks_num,1,1);
	int threads_num;

	if(blocks_num==1)
		threads_num=num_sums/2;
	else
		threads_num=MAX_NUM_OF_THREADS_PER_BLOCK;

	if(!threads_num) return;

	dim3 threads(threads_num,1,1);
	printf("first of the accumulating functions: blocks=%d threads in each one=%d\n",blocks_num,threads_num);
	accumulate_sums<<<grid,threads,4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK>>> (d_sums,2);
	cutilCheckMsg("accumulate_sums");

	if(blocks_num==1) return;

	dim3 grid2(1,1,1);
	dim3 threads2(blocks_num/2,1,1);
	int offset=MAX_NUM_OF_THREADS_PER_BLOCK*2;
	printf("offset=%d threads to sum of sums = %d \n",offset,threads2.x);
	accumulate_sum_of_sums<<<grid2,threads2,4*sizeof(int)*threads2.x>>> (d_sums,2,offset);
	cutilCheckMsg("accumulate_sum_of_sums");
}

void quicksort(elem* d_elems,sum* d_sums,int num_elements,int n,int num_elements_per_block,int num_blocks,int num_blocks2){

	dim3  grid(num_blocks, 1, 1); // 
//	dim3  threads(num_elements/2, 1, 1);
	dim3  threads(MAX_NUM_OF_THREADS_PER_BLOCK, 1, 1);

	int num_threads2=num_blocks2/2;
	num_threads2+=num_blocks2 & 1;
	while(num_threads2>MAX_NUM_OF_THREADS_PER_BLOCK)
		if(num_threads2 & 1){
			num_threads2>>=1;
			++num_threads2;
		}else
			num_threads2>>=1;

	dim3 grid2(1,1,1);
	dim3 threads2(num_threads2,1,1);

	printf("mikki: threads=%d elems=%d elems_per_block%d blocks=%d blocks2=%d n=%d\n",num_threads2,num_elements,num_elements_per_block,num_blocks,num_blocks2,n);

	// zakomentowane na jakis czas:
	/*
	check_order<<< grid, threads, sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements,num_elements_per_block,num_blocks,num_blocks2);

	check_order2<<< grid2, threads2,  sizeof(int)*num_threads2 >>>
		(d_sums,num_blocks2);
		*/

/*	
	make_pivots<<< grid, threads, sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements,num_elements_per_block,num_blocks,num_blocks2);
		*/
	make_pivots<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK);

	up_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);

	down_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);
	cutilCheckMsg("down_sweep");

	make_pivots2<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK,num_blocks2);

	make_offsets<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK);

	up_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);

	down_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);
	cutilCheckMsg("down_sweep");

	make_offsets2<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK,num_blocks2);

	make_idowns<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK);

	up_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);

	down_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);
	cutilCheckMsg("down_sweep");

	make_idowns2<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK,num_blocks2);

	make_iup1s<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK);

	up_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);

	down_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);
	cutilCheckMsg("down_sweep");

	make_iup1s2<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK,num_blocks2);

	printf("threads=%d\n",threads.x);
	printf("num_blocks=%d\n",num_blocks);
	make_iup2s<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK, num_blocks);

	printf("num_blocks2=%d num_elements_per_block=%d\n",num_blocks2,num_elements_per_block);
	up_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);

	printf("numBLOCKS=%d num_el=%d\n",num_blocks2,num_elements_per_block);

	down_sweep_for_sum(d_sums,num_blocks2,num_elements_per_block);
	cutilCheckMsg("down_sweep");

	make_iup2s2<<< grid, threads, 4*sizeof(int)*MAX_NUM_OF_THREADS_PER_BLOCK >>>
		(d_elems, d_sums, num_elements_per_block/MAX_NUM_OF_THREADS_PER_BLOCK,num_blocks2,num_blocks);
	
	cutilCheckMsg("move_elems1");
	move_elems1<<< grid, threads >>> (d_elems, 2, num_elements);
	
	cutilCheckMsg("move_elems2");
	move_elems2<<< grid, threads >>> (d_elems, 2, num_elements);
	
	cutilCheckMsg("move_elems3");
	move_elems3<<< grid, threads >>> (d_elems, 2, num_elements);
/*	
		*/
}

void
runTest( int argc, char** argv) 
{
	/*
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );
*/
	unsigned int num_elements = NUM_OF_ELEMENTS;

	int num_elements_per_thread=2;
	int num_threads=num_elements/num_elements_per_thread;
	if(num_elements & 1)
		num_threads+=1;

	while(num_threads>MAX_NUM_OF_THREADS_PER_BLOCK*MAX_NUM_OF_BLOCKS){
		int a=1 & num_threads; // reszta z dzielenia przez 2
		num_threads>>=1;
		num_threads+=a;
		num_elements_per_thread<<=1;
	}

	unsigned int num_blocks=num_threads/MAX_NUM_OF_THREADS_PER_BLOCK;
	if(num_blocks*MAX_NUM_OF_THREADS_PER_BLOCK!=num_threads)
		num_blocks+=1;

	const unsigned int n = num_blocks*MAX_NUM_OF_THREADS_PER_BLOCK*num_elements_per_thread;
	unsigned int timer;
	cutilCheckError( cutCreateTimer(&timer));

	tab* table;

	int num_blocks2=1;
	while(num_blocks2<num_blocks) num_blocks2<<=1;
	
	table=make_tab(n,num_blocks2);

	elem* d_elems;
	sum* d_sums;
	  
	// initialize the input data on the host to be integer values
	// between 0 and 1000
	srand(time(NULL));
	printf("elems(n=%d) to be sorted:\n",num_elements);
	for( unsigned int i = 0; i < num_elements; ++i) 
	{
		int elval = 1000*(rand()/(float)RAND_MAX);
		if(rand() & 1) elval*=-1;

		/*
		// UWAGA: ponizej jest kod do testow:
		if(rand() & 1) elval=1;
		else elval=0;
		*/

		table->elems[i].val = elval;
		table->elems[i].seg_flag2=0;
		table->elems[i].pivot=0;
		printf(" %d ",table->elems[i].val);
	}
	printf(" ;\n");
	table->elems[0].seg_flag2=1;
	table->elems[0].val=-198;
	table->elems[1].val=-975;
	table->elems[2].val=62;
	table->elems[3].val=-234;
	for(unsigned int i=num_elements;i<n;++i){
		table->elems[i].val=INT_MAX;
		table->elems[i].seg_flag2=1;
		table->elems[i].pivot=0;
	}
	for(int i=0;i<num_blocks2;++i)
		table->sums[i].val=0;

	cutilSafeCall( hipMalloc( (void**) &d_elems, table->n*sizeof(elem)));
	cutilSafeCall( hipMalloc( (void**) &d_sums, num_blocks2*sizeof(sum)));
	cutilSafeCall( hipMemcpy( d_elems, table->elems, table->n*sizeof(elem), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy( d_sums, table->sums, num_blocks2*sizeof(sum), hipMemcpyHostToDevice) );

//	const unsigned int num_threads_per_block=MAX_NUM_OF_THREADS_PER_BLOCK;
	const unsigned int num_elements_per_block=n/num_blocks;


	// TO BE CHANGED
//	const unsigned int shared_mem_size=16384;//NUM_OF_ARRAYS_PER_BLOCK*sizeof(float)*num_threads_per_block*2;

	// make sure there are no CUDA errors before we start
	cutilCheckMsg("Kernel execution failed");

	printf("Running parallel quicksort for %d elements (n=%d)\n", num_elements,n);
  
	unsigned int numIterations = 1;
	
	printf("pbGPUQuicksort with params:\n- blocks=%d,\n- elements=%d,\n- elements2thread=%d\n"
			,num_blocks,num_elements,num_elements_per_thread);
	cutilCheckError(cutStartTimer(timer));
	for (unsigned int i = 0; i < numIterations; ++i)
	{
		quicksort(d_elems,d_sums,num_elements,n,num_elements_per_block,num_blocks,num_blocks2);
	}
	hipDeviceSynchronize();
	cutilCheckError(cutStopTimer(timer));
	printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
//	printf("CPU time: %f ms\n\n", time);
	cutResetTimer(timer);

	// check for any errors
	cutilCheckMsg("Kernel execution failed");

	cutilSafeCall(hipMemcpy( table->elems, d_elems,table->n*sizeof(elem),hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy( table->sums, d_sums,num_blocks2*sizeof(sum),hipMemcpyDeviceToHost));
	for( unsigned int i = 0; i < n; ++i)
		printf("val[%d] = %d seg_flag2 = %d pivot[%d] = %d offset=%d idown=%d iup=%d iup2=%d flag=%d\n",i,table->elems[i].val,table->elems[i].seg_flag2,i,table->elems[i].pivot,table->elems[i].offset,table->elems[i].idown,table->elems[i].iup1,table->elems[i].iup2,table->elems[i].seg_flag);
	for( unsigned int i = 0; i < num_blocks2; ++i)
		printf("sum[%d] = %d seg_flag=%d\n",i,table->sums[i].val,table->sums[i].seg_flag);
//	printf("sum[%d] = %d\n",0,table->sums[0].val);
//	printf("thread[%d] = %d\n",n-1,table->elems[n-1].val);
	printf("\nAuthor: Paweł Baran. e-mail: shatov33@gmail.com .\n");

	// cleanup memory
	printf("a3\n");
	cutilSafeCall(hipFree(d_elems));
	printf("a4\n");
	cutilSafeCall(hipFree(d_sums));
	printf("a5\n");
//	free_tab(table);
	printf("a6\n");
	cutilCheckError(cutDeleteTimer(timer));
	printf("a7\n");

	hipDeviceReset();
}
